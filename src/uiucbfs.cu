#include "hip/hip_runtime.h"
# include "ubfsfunc.h"
# include "uiucbfs.cuh"
# include "timer.h"
// int3: next_wf, local-Q
namespace ubfs
{

void parWavefront(std::vector<int> &srcNode,
		Graph<AdjacentNode> &cuGraph,
		std::vector<int> &distances,
		std::vector<int> &cameFrom,
		int destination)
{
	bool *d_is_end;
	bool h_is_end=false;
	CUDA_ALLOC_DEV_MEM(&d_is_end,sizeof(bool));
	CUDA_MEMCPY_H2D(d_is_end,&h_is_end,sizeof(bool));

	thrust::device_vector<int> rounds(1);
	rounds[0]=0;
	thrust::device_vector<int> d_cameFrom(cuGraph.numVertices);
	thrust::copy(cameFrom.begin(),cameFrom.end(),d_cameFrom.begin());

	thrust::device_vector<Edge> d_graph_node(cuGraph.numVertices);
	for(int i=0;i<cuGraph.numVertices;i++)
	{
		int2 tmp;
		tmp.x=cuGraph.edgesOffset[i];
		tmp.y=cuGraph.edgesSize[i];
		d_graph_node[i]=tmp;
	}
	thrust::device_vector<Node> d_graph_edge(cuGraph.numEdges);
	for(int i=0;i<cuGraph.numEdges;i++)
	{
		int2 tmp;
		tmp.x=cuGraph.adjacencyList[i].terminalVertex;
		tmp.y=cuGraph.adjacencyList[i].weight;
		d_graph_edge[i]=tmp;
	}

	using thrust::raw_pointer_cast;
	  //bind the texture memory with global memory
	  hipBindTexture(0,g_graph_node_ref,raw_pointer_cast(&d_graph_node[0]), sizeof(Node)*cuGraph.numVertices);
	  hipBindTexture(0,g_graph_edge_ref,raw_pointer_cast(&d_graph_edge[0]), sizeof(Edge)*cuGraph.numEdges);


	 int* d_color;
	  hipMalloc((void**) &d_color, sizeof(int)*cuGraph.numVertices);
	  int* d_cost;
	  hipMalloc((void**) &d_cost, sizeof(int)*cuGraph.numVertices);
	  int * d_q1;
	  int * d_q2;
	  hipMalloc((void**) &d_q1, sizeof(int)*cuGraph.numVertices);
	  hipMalloc((void**) &d_q2, sizeof(int)*cuGraph.numVertices);
	  int * tail;
	  hipMalloc((void**) &tail, sizeof(int));
	  int *front_cost_d;
	  hipMalloc((void**) &front_cost_d, sizeof(int));
	  CUDA_DEV_MEMSET(d_color,WHITE,sizeof(int)*cuGraph.numVertices);
//	  CUDA_DEV_MEMSET(d_cost,1000,sizeof(int)*cuGraph.numVertices);
//	  hipMemcpy(&d_cost[srcNode[0]],0,sizeof(int),hipMemcpyHostToDevice);
	  CUDA_MEMCPY_H2D(d_cost,&distances[0], sizeof(int)*cuGraph.numVertices);

	  int h_top=1;
	  hipMemcpy(tail,&h_top,sizeof(int),hipMemcpyHostToDevice);


	  hipMemcpy( &d_q1[0], &srcNode[0], sizeof(int), hipMemcpyHostToDevice);



	  //whether or not to adjust "k", see comment on "BFS_kernel_multi_blk_inGPU" for more details
	  int * switch_kd;
	  hipMalloc((void**) &switch_kd, sizeof(int));
	  int * num_td;//number of threads
	  hipMalloc((void**) &num_td, sizeof(int));

	  //whether to stay within a kernel, used in "BFS_kernel_multi_blk_inGPU"
	  bool *stay;
	  hipMalloc( (void**) &stay, sizeof(bool));
	  int switch_k;

	  //max number of frontier nodes assigned to a block
	  int * max_nodes_per_block_d;
	  hipMalloc( (void**) &max_nodes_per_block_d, sizeof(int));
	  int *global_kt_d;
	  hipMalloc( (void**) &global_kt_d, sizeof(int));
	  CUDA_DEV_MEMSET(global_kt_d,0,sizeof(int));

	  int h_overflow = 0;
	  int *d_overflow;
	  hipMalloc((void**) &d_overflow, sizeof(int));
	  hipMemcpy(d_overflow, &h_overflow, sizeof(int), hipMemcpyHostToDevice);

	  int num_t;//number of threads
	  int k=0;//BFS level index
	  int num_of_blocks;
	  int num_of_threads_per_block;


	  	GpuTimer tm1;
	  	tm1.Start();
	  do
	   {
	     hipMemcpy( &num_t, tail, sizeof(int), hipMemcpyDeviceToHost);
	     CUDA_DEV_MEMSET(tail,0,sizeof(int));

	     if(num_t == 0){//frontier is empty
	       hipFree(stay);
	       hipFree(switch_kd);
	       hipFree(num_td);
	       break;
	     }

	     num_of_blocks = 1;
	     num_of_threads_per_block = num_t;
	     if(num_of_threads_per_block <NUM_BIN)
	       num_of_threads_per_block = NUM_BIN;
	     if(num_t>MAX_THREADS_PER_BLOCK)
	     {
	       num_of_blocks = (int)ceil(num_t/(double)MAX_THREADS_PER_BLOCK);
	       num_of_threads_per_block = MAX_THREADS_PER_BLOCK;
	     }
	     if(num_of_blocks == 1)//will call "BFS_in_GPU_kernel"
	       num_of_threads_per_block = MAX_THREADS_PER_BLOCK;
	     if(num_of_blocks >1 && num_of_blocks <= NUM_SM)// will call "BFS_kernel_multi_blk_inGPU"
	       num_of_blocks = NUM_SM;

	     //assume "num_of_blocks" can not be very large
	     dim3  grid( num_of_blocks, 1, 1);
	     dim3  threads( num_of_threads_per_block, 1, 1);

	     if(k%2 == 0){
	       if(num_of_blocks == 1){
	         BFS_in_GPU_kernel<<< grid, threads >>>(d_q1,d_q2, raw_pointer_cast(&d_graph_node[0]),
	             raw_pointer_cast(&d_graph_edge[0]), d_color, d_cost,num_t , tail,GRAY0,k,d_overflow,
	             destination,d_is_end, raw_pointer_cast(&rounds[0]),raw_pointer_cast(&d_cameFrom[0]));
	       }
	       else if(num_of_blocks <= NUM_SM){
	         (hipMemcpy(num_td,&num_t,sizeof(int),
	                     hipMemcpyHostToDevice));
	         BFS_kernel_multi_blk_inGPU
	           <<< grid, threads >>>(d_q1,d_q2, raw_pointer_cast(&d_graph_node[0]),
	               raw_pointer_cast(&d_graph_edge[0]), d_color, d_cost, num_td, tail,GRAY0,k,
	               switch_kd, max_nodes_per_block_d, global_kt_d,d_overflow,
	               destination,d_is_end,raw_pointer_cast(&d_cameFrom[0]));
	         (hipMemcpy(&switch_k,switch_kd, sizeof(int),
	                     hipMemcpyDeviceToHost));
	         if(!switch_k){
	           k--;
	         }
	       }
	       else{
	         BFS_kernel<<< grid, threads >>>(d_q1,d_q2, raw_pointer_cast(&d_graph_node[0]),
	             raw_pointer_cast(&d_graph_edge[0]), d_color, d_cost, num_t, tail,GRAY0,k,d_overflow,
	             destination,d_is_end,raw_pointer_cast(&d_cameFrom[0]));
	       }
	     }
	     else{
	       if(num_of_blocks == 1){
	         BFS_in_GPU_kernel<<< grid, threads >>>(d_q2,d_q1, raw_pointer_cast(&d_graph_node[0]),
	             raw_pointer_cast(&d_graph_edge[0]), d_color, d_cost, num_t, tail,GRAY1,k,d_overflow,
	             destination,d_is_end,raw_pointer_cast(&rounds[0]),raw_pointer_cast(&d_cameFrom[0]));
	       }
	       else if(num_of_blocks <= NUM_SM){
	         (hipMemcpy(num_td,&num_t,sizeof(int),
	                     hipMemcpyHostToDevice));
	         BFS_kernel_multi_blk_inGPU
	           <<< grid, threads >>>(d_q2,d_q1, raw_pointer_cast(&d_graph_node[0]),
	               raw_pointer_cast(&d_graph_edge[0]), d_color, d_cost, num_td, tail,GRAY1,k,
	               switch_kd, max_nodes_per_block_d, global_kt_d,d_overflow,
	               destination,d_is_end,raw_pointer_cast(&d_cameFrom[0]));
	         (hipMemcpy(&switch_k,switch_kd, sizeof(int),
	                     hipMemcpyDeviceToHost));
	         if(!switch_k){
	           k--;
	         }
	       }
	       else{
	         BFS_kernel<<< grid, threads >>>(d_q2,d_q1, raw_pointer_cast(&d_graph_node[0]),
	             raw_pointer_cast(&d_graph_edge[0]), d_color, d_cost, num_t, tail, GRAY1,k,d_overflow,
	             destination,d_is_end,raw_pointer_cast(&d_cameFrom[0]));
	       }
	     }
	     k++;
	     hipMemcpy(&h_overflow, d_overflow, sizeof(int), hipMemcpyDeviceToHost);
	     if(h_overflow) {
	       printf("Error: local queue was overflow. Need to increase W_LOCAL_QUEUE\n");
	       return;
	     }
	     // copy is end d 2 h
	     CUDA_MEMCPY_D2H(&h_is_end,d_is_end,sizeof(bool));
	     if(h_is_end)
	     {
	    	 // copy dist d 2 h
	    	 CUDA_MEMCPY_D2H(&distances[0],d_cost, sizeof(int)*cuGraph.numVertices);
	    	 int rnd=rounds[0];
	    	 printf("final distance is %d , levels  is %d \n",distances[destination],rnd);
	    	break;
	     }
	   } while(1);
	  tm1.Stop();
	  std::cout<<"total time is "<<float(tm1.Elapsed())<<" ms"<<std::endl;
	   hipDeviceSynchronize();
	   printf("GPU kernel done\n");

	   // copy result from device to host
//	   hipMemcpy(&distances[0], d_cost, sizeof(int)*cuGraph.numVertices, hipMemcpyDeviceToHost);
		// retrace
		std::vector<int> search_path;
		int ptOnPath=destination;
		while(d_cameFrom[ptOnPath]!= ptOnPath)
		{
			search_path.push_back(ptOnPath);
			ptOnPath= d_cameFrom[ptOnPath];
		}
	//	std::reverse(search_path.begin(),search_path.end());
		for(auto ptr:search_path)
		{
			std::cout<<ptr<<",";
		}
	   hipUnbindTexture(g_graph_node_ref);
	   hipUnbindTexture(g_graph_edge_ref);

	   hipFree(d_color);
	   hipFree(d_cost);
	   hipFree(tail);
	   hipFree(front_cost_d);


}

}
